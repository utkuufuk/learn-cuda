#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"
#include "utils.h"

const int N= 1024;	// matrix size will be NxN
const int K= 32;	// TODO, set K to the correct value and tile size will be KxK


// to be launched with one thread per element, in (tilesize)x(tilesize) threadblocks
// thread blocks read & write tiles, in coalesced fashion
// adjacent threads read adjacent input elements, write adjacent output elmts
__global__ void 
transpose_parallel_per_element_tiled(float in[], float out[])
{
    int in_corner_i = blockIdx.x * K;
    int in_corner_j = blockIdx.y * K;
    
    int out_corner_j = blcokIdx.y * K;
    int out_corner_i = blockIdx.x * K;
    
    int x = threadIdx.x;
    int y = threadIdx.y;
    
    __shared__ float tile[K][K];

	tile[y][x] = in[(in_corner_i + x) + (in_corner_j + j) * N];
	__syncthreads();
	
	out[(out_corner_i + x) + (out_corner_j + y) * N] = tile[x][y];
}

// The following functions and kernels are for your references
void 
transpose_CPU(float in[], float out[])
{
	for(int j=0; j < N; j++)
    	for(int i=0; i < N; i++)
      		out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched on a single thread
__global__ void 
transpose_serial(float in[], float out[])
{
	for(int j=0; j < N; j++)
		for(int i=0; i < N; i++)
			out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched with one thread per row of output matrix
__global__ void 
transpose_parallel_per_row(float in[], float out[])
{
	int i = threadIdx.x;

	for(int j=0; j < N; j++)
		out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched with one thread per element, in KxK threadblocks
// thread (x,y) in grid writes element (i,j) of output matrix 
__global__ void 
transpose_parallel_per_element(float in[], float out[])
{
	int i = blockIdx.x * K + threadIdx.x;
	int j = blockIdx.y * K + threadIdx.y;

	out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

int main(int argc, char **argv)
{
	int numbytes = N * N * sizeof(float);

	float *in = (float *) malloc(numbytes);
	float *out = (float *) malloc(numbytes);
	float *gold = (float *) malloc(numbytes);

	fill_matrix(in, N);
	transpose_CPU(in, gold);

	float *d_in, *d_out;

	hipMalloc(&d_in, numbytes);
	hipMalloc(&d_out, numbytes);
	hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);

	GpuTimer timer;

/*  
 * Now time each kernel and verify that it produces the correct result.
 *
 * To be really careful about benchmarking purposes, we should run every kernel once
 * to "warm" the system and avoid any compilation or code-caching effects, then run 
 * every kernel 10 or 100 times and average the timings to smooth out any variance. 
 * But this makes for messy code and our goal is teaching, not detailed benchmarking.
 */

	dim3 blocks(N,N);	//TODO, you need to set the proper blocks per grid
	dim3 threads(K,K);	//TODO, you need to set the proper threads per block

	timer.Start();
	transpose_parallel_per_element_tiled<<<blocks,threads>>>(d_in, d_out);
	timer.Stop();
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element_tiled %dx%d: %g ms.\nVerifying ...%s\n", 
		   K, K, timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

	hipFree(d_in);
	hipFree(d_out);
}