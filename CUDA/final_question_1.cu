#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "compare.h"
#include "gputimer.h"

__global__ void smooth(float * v_new, const float * v) 
{
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;
    int myLeftIdx = (myIdx == 0) ? 0 : myIdx - 1;
    int myRightIdx = (myIdx == (numThreads - 1)) ? numThreads - 1 : myIdx + 1;
    float myElt = v[myIdx];
    float myLeftElt = v[myLeftIdx];
    float myRightElt = v[myRightIdx];
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
}

__global__ void smooth_shared(float * v_new, const float * v) 
{
    extern __shared__ float s[];
    
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int numElements = blockDim.x * gridDim.x;

    s[threadIdx.x + 1] = v[myIdx];

    if (myIdx == 0)
    {
        s[0] = v[0];
    }
    else if (threadIdx.x == 0)
    {
        s[0] = v[myIdx - 1];
    }

    if (myIdx == numElements - 1)
    {
        s[257] = v[numElements - 1];
    }
    else if (threadIdx.x == blockDim.x - 1)
    {
        s[257] = v[myIdx + 1];
    }
    __syncthreads();

    float myElt = s[threadIdx.x + 1];
    float myLeftElt = s[threadIdx.x];
    float myRightElt = s[threadIdx.x + 2];
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
}

int main(int argc, char **argv)
{
    const int ARRAY_SIZE = 4096;
    const int BLOCK_SIZE = 256;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
    const int BLOCK_BYTES = (BLOCK_SIZE + 2) * sizeof(float);

    // generate the input array on the host
    float h_in[ARRAY_SIZE];
    float h_cmp[ARRAY_SIZE];
    float h_out[ARRAY_SIZE];
    float h_out_shared[ARRAY_SIZE];
    
    for (int i = 0; i < ARRAY_SIZE; i++) 
    {
        // generate random float in [0, 1]
        h_in[i] = (float) random() / (float) RAND_MAX;
    }
    
    for (int i = 0; i < ARRAY_SIZE; i++) 
    {
        h_cmp[i] = (0.25f * h_in[(i == 0) ? 0 : i - 1] + 0.50f * h_in[i] +
                    0.25f * h_in[(i == (ARRAY_SIZE - 1)) ? ARRAY_SIZE - 1 : i + 1]);
    }

    // declare GPU memory pointers
    float * d_in, * d_out, * d_out_shared;

    // allocate GPU memory
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);
    hipMalloc((void **) &d_out_shared, ARRAY_BYTES);

    // transfer the input array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice); 

    // launch the reference kernel
    GpuTimer timer;
    timer.Start();
    smooth<<<ARRAY_SIZE / BLOCK_SIZE, BLOCK_SIZE>>>(d_out, d_in);
    timer.Stop();

    printf("Reference code executed in %g ms\n", timer.Elapsed());  

    // launch the student kernel
    timer.Start();
    //smooth<<<ARRAY_SIZE / BLOCK_SIZE, BLOCK_SIZE>>>(d_out_shared, d_in);
    smooth_shared<<<ARRAY_SIZE / BLOCK_SIZE, BLOCK_SIZE, BLOCK_BYTES>>>(d_out_shared, d_in);
    timer.Stop();

    printf("Your code executed in %g ms\n", timer.Elapsed());  

    // copy back the result from GPU
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(h_out_shared, d_out_shared, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // testing for correctness
    compare(h_in, h_out, h_out_shared, h_cmp, ARRAY_SIZE);

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_shared);
}
