#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

const int BLOCK_ROWS = 32;
const int BLOCK_COLS = 32;
const int MAX_THREADS_PER_BLOCK = 1024;

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;
float         *d_redTemp, *d_greenTemp, *d_blueTemp;

    __global__
void gaussianBlur(const unsigned char* const inputChannel,
                  float* const outputChannel,
                  int numRows,   
                  int numCols,
                  int numThreadMatrices,
                  const float* const filter, 
                  const int filterWidth)
{       
    // calculate the center pixel location of the thread 
    int centerColIndex = blockIdx.x * numThreadMatrices + threadIdx.x;
    int centerRowIndex = blockIdx.y;
    int centerPixelIndex = centerRowIndex * numCols + centerColIndex;

    // return if the center index is out of bounds
    if (centerColIndex >= numCols || centerRowIndex >= numRows)
    {
        return;
    }

    // calculate the corresponding filter coefficient index for this thread
    int rowOffset = (-filterWidth / 2) + threadIdx.z; 
    int colOffset = (-filterWidth / 2) + threadIdx.y;
    int filterIndex = threadIdx.z * filterWidth + threadIdx.y;

    // calculate the mapped rows and columns of each thread 
    int threadRowIndex = min(max(centerRowIndex + rowOffset, 0), static_cast<int>(numRows - 1));
    int threadColIndex = min(max(centerColIndex + colOffset, 0), static_cast<int>(numCols - 1));
    int threadPixelIndex = threadRowIndex * numCols + threadColIndex;

    // atomically update the weighted sum for the center pixetl
    atomicAdd(&outputChannel[threadPixelIndex], inputChannel[centerPixelIndex] * filter[filterIndex]);
}

__global__
void copy(unsigned char* const outputChannel, float* const inputChannel, int numRows, int numCols)
{   
    const int2 threadIndex2D = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                         blockIdx.y * blockDim.y + threadIdx.y);
    const int index = threadIndex2D.y * numCols + threadIndex2D.x;

    // avoid accessing the memory outside the image by having any threads mapped there return early
    if (threadIndex2D.x >= numCols || threadIndex2D.y >= numRows)
    {
        return;
    }
    outputChannel[index] = inputChannel[index];
}

    __global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{   
    const int2 threadIndex2D = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                         blockIdx.y * blockDim.y + threadIdx.y);
    const int index = threadIndex2D.y * numCols + threadIndex2D.x;

    // avoid accessing the memory outside the image by having any threads mapped there return early
    if (threadIndex2D.x >= numCols || threadIndex2D.y >= numRows)
    {
        return;
    }
    uchar4 rgba = inputImageRGBA[index];                
    redChannel[index] = rgba.x;
    greenChannel[index] = rgba.y;
    blueChannel[index] = rgba.z;
}

    __global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
    const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                         blockIdx.y * blockDim.y + threadIdx.y);
    const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

    // avoid accessing the memory outside the image by having any threads mapped there return early
    if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    {
        return;
    }
    unsigned char red   = redChannel[thread_1D_pos];
    unsigned char green = greenChannel[thread_1D_pos];
    unsigned char blue  = blueChannel[thread_1D_pos];

    //Alpha should be 255 for no transparency
    uchar4 outputPixel = make_uchar4(red, green, blue, 255);
    outputImageRGBA[thread_1D_pos] = outputPixel;
}

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{
    checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
    checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
    checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

    checkCudaErrors(hipMalloc(&d_redTemp, sizeof(int) * numRowsImage * numColsImage));
    checkCudaErrors(hipMalloc(&d_greenTemp, sizeof(int) * numRowsImage * numColsImage));
    checkCudaErrors(hipMalloc(&d_blueTemp, sizeof(int) * numRowsImage * numColsImage));

    checkCudaErrors(hipMemset(d_redTemp, 0, sizeof(int) * numRowsImage * numColsImage));
    checkCudaErrors(hipMemset(d_greenTemp, 0, sizeof(int) * numRowsImage * numColsImage));
    checkCudaErrors(hipMemset(d_blueTemp, 0, sizeof(int) * numRowsImage * numColsImage));

    size_t filterMemSize = filterWidth * filterWidth * sizeof(float);
    checkCudaErrors(hipMalloc(&d_filter, filterMemSize));
    checkCudaErrors(hipMemcpy(d_filter, h_filter, filterMemSize, hipMemcpyHostToDevice));
}

void your_gaussian_blur(const uchar4 * const h_inputRGBA, 
                        uchar4 * const d_inputRGBA,
                        uchar4* const d_outputRGBA, 
                        const size_t numRows, 
                        const size_t numCols,
                        unsigned char *d_redBlurred, 
                        unsigned char *d_greenBlurred, 
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{
    // set the thread and block sizes for kernels that seperate and recombine the channels
    const dim3 channelThreads(BLOCK_COLS, BLOCK_ROWS);
    const dim3 channelBlocks(1 + (numCols / channelThreads.x), 1 + (numRows / channelThreads.y));

    // set the thread and block sizes for the blurring kernel
    int threadsPerBlurBlock = MAX_THREADS_PER_BLOCK / (filterWidth * filterWidth);
    const dim3 blurThreads(threadsPerBlurBlock, filterWidth, filterWidth);
    const dim3 blurBlocks((numCols / threadsPerBlurBlock) + 1, numRows);

    // print useful information
    printf("Image size: %dx%d\n", numCols, numRows);
    printf("Threads for channel kernels: %dx%d\nBlocks for channel kernels: %dx%d\n",
            channelThreads.x, channelThreads.y, channelBlocks.x, channelBlocks.y);
    printf("Threads for blurring a channel: %dx%dx%d\nBlocks for blurring a channel: %dx%d\n",
            blurThreads.x, blurThreads.y, blurThreads.z, blurBlocks.x, blurBlocks.y);

    // separate the color channels
    separateChannels<<<channelBlocks, channelThreads>>>
                    (d_inputRGBA, numRows, numCols, d_red, d_green, d_blue);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    // blur the red channel 
    gaussianBlur<<<blurBlocks, blurThreads, threadsPerBlurBlock>>>
                (d_red, d_redTemp, numRows, numCols, threadsPerBlurBlock, d_filter, filterWidth);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    // blur the green channel
    gaussianBlur<<<blurBlocks, blurThreads, threadsPerBlurBlock>>>
                (d_green, d_greenTemp, numRows, numCols, threadsPerBlurBlock, d_filter, filterWidth);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    // blur the blue channel
    gaussianBlur<<<blurBlocks, blurThreads, threadsPerBlurBlock>>>
                (d_blue, d_blueTemp, numRows, numCols, threadsPerBlurBlock, d_filter, filterWidth);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    copy<<<channelBlocks, channelThreads>>>(d_redBlurred, d_redTemp, numRows, numCols);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    copy<<<channelBlocks, channelThreads>>>(d_greenBlurred, d_greenTemp, numRows, numCols);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    copy<<<channelBlocks, channelThreads>>>(d_blueBlurred, d_blueTemp, numRows, numCols);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    // recombine the blurred channels
    recombineChannels<<<channelBlocks, channelThreads>>>
                    (d_redBlurred, d_greenBlurred, d_blueBlurred, d_outputRGBA, numRows, numCols);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());
}

void cleanup() 
{
    checkCudaErrors(hipFree(d_red));
    checkCudaErrors(hipFree(d_green));
    checkCudaErrors(hipFree(d_blue));
    checkCudaErrors(hipFree(d_redTemp));
    checkCudaErrors(hipFree(d_greenTemp));
    checkCudaErrors(hipFree(d_blueTemp));
    checkCudaErrors(hipFree(d_filter));
}
