#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

__global__
void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, 
		           int numCols,
                   const float* const filter, 
        		   const int filterWidth)
{	
	const int2 threadIndex2D = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                         blockIdx.y * blockDim.y + threadIdx.y);
    const int index = threadIndex2D.y * numCols + threadIndex2D.x;
    
    if (threadIndex2D.x >= numCols || threadIndex2D.y >= numRows)
    {
        return;
    }

    float color = 0.0f;
    
    for (int filterRowIndex = 0; filterRowIndex < filterWidth; filterRowIndex++) 
    {
        for (int filterColIndex = 0; filterColIndex < filterWidth; filterColIndex++) 
        {
            int colIndex = threadIndex2D.x + filterColIndex - filterWidth/2;
            int rowIndex = threadIndex2D.y + filterRowIndex - filterWidth/2;
            colIndex = min(max(colIndex, 0), numCols - 1);
            rowIndex = min(max(rowIndex, 0), numRows - 1);
            float filter_value = filter[filterRowIndex*filterWidth + filterColIndex];
            color += filter_value * static_cast<float>(inputChannel[rowIndex * numCols + colIndex]);
        }
    }
    outputChannel[index] = color;
}

__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{	
	const int2 threadIndex2D = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                         blockIdx.y * blockDim.y + threadIdx.y);

	const int index = threadIndex2D.y * numCols + threadIndex2D.x;

    if (threadIndex2D.x >= numCols || threadIndex2D.y >= numRows)
    {
        return;
    }
	uchar4 rgba = inputImageRGBA[index];				
	redChannel[index] = rgba.x;
	greenChannel[index] = rgba.y;
	blueChannel[index] = rgba.z;
}

__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
    const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                       blockIdx.y * blockDim.y + threadIdx.y);

    const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

    //make sure we don't try and access memory outside the image by having any threads mapped there return early
    if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    {
        return;
    }
    unsigned char red   = redChannel[thread_1D_pos];
    unsigned char green = greenChannel[thread_1D_pos];
    unsigned char blue  = blueChannel[thread_1D_pos];

    //Alpha should be 255 for no transparency
    uchar4 outputPixel = make_uchar4(red, green, blue, 255);
    outputImageRGBA[thread_1D_pos] = outputPixel;
}

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{
    checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
    checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
    checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

    checkCudaErrors(hipMalloc(&d_filter,  sizeof(float) * filterWidth * filterWidth));
    checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));
}

void your_gaussian_blur(const uchar4* const h_inputImageRGBA, 
	            		uchar4* const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, 
               			const size_t numRows, 
			            const size_t numCols,
                        unsigned char *d_redBlurred, 
                        unsigned char *d_greenBlurred, 
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{
    const dim3 blockSize(32, 32);
    const dim3 gridSize(numCols / blockSize.x + 1, numRows / blockSize.y + 1);

    separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA, numRows, numCols, d_red, d_green, d_blue);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    gaussian_blur<<<gridSize, blockSize>>>(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    gaussian_blur<<<gridSize, blockSize>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    gaussian_blur<<<gridSize, blockSize>>>(d_blue, d_blueBlurred, numRows, numCols, d_filter, filterWidth);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    recombineChannels<<<gridSize, blockSize>>>(d_redBlurred, d_greenBlurred, d_blueBlurred, d_outputImageRGBA, numRows, numCols);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());
}

void cleanup() 
{
    checkCudaErrors(hipFree(d_red));
    checkCudaErrors(hipFree(d_green));
    checkCudaErrors(hipFree(d_blue));
    checkCudaErrors(hipFree(d_filter));
}
