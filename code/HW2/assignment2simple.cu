#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

#define BLOCK_ROWS 32
#define BLOCK_COLS 32

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

__global__
void gaussianBlur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, 
		           int numCols,
                   const float* const filter, 
        		   const int filterWidth)
{	
	const int2 threadIndex2D = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                         blockIdx.y * blockDim.y + threadIdx.y);
    const int index = threadIndex2D.y * numCols + threadIndex2D.x;
    
    if (threadIndex2D.x >= numCols || threadIndex2D.y >= numRows)
    {
        return;
    }
    float color = 0.0f;
    
    for (int filterRowIndex = 0; filterRowIndex < filterWidth; filterRowIndex++) 
    {
        for (int filterColIndex = 0; filterColIndex < filterWidth; filterColIndex++) 
        {
            int colIndex = threadIndex2D.x + filterColIndex - filterWidth/2;
            int rowIndex = threadIndex2D.y + filterRowIndex - filterWidth/2;
            colIndex = min(max(colIndex, 0), numCols - 1);
            rowIndex = min(max(rowIndex, 0), numRows - 1);
            float filter_value = filter[filterRowIndex*filterWidth + filterColIndex];
            color += filter_value * static_cast<float>(inputChannel[rowIndex * numCols + colIndex]);
        }
    }
    outputChannel[index] = color;
}

__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{	
	const int2 threadIndex2D = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                         blockIdx.y * blockDim.y + threadIdx.y);

	const int index = threadIndex2D.y * numCols + threadIndex2D.x;

    if (threadIndex2D.x >= numCols || threadIndex2D.y >= numRows)
    {
        return;
    }
	uchar4 rgba = inputImageRGBA[index];				
	redChannel[index] = rgba.x;
	greenChannel[index] = rgba.y;
	blueChannel[index] = rgba.z;
}

__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
    const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                       blockIdx.y * blockDim.y + threadIdx.y);

    const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

    // avoid accessing the memory outside the image by having any threads mapped there return early
    if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    {
        return;
    }
    unsigned char red   = redChannel[thread_1D_pos];
    unsigned char green = greenChannel[thread_1D_pos];
    unsigned char blue  = blueChannel[thread_1D_pos];

    //Alpha should be 255 for no transparency
    uchar4 outputPixel = make_uchar4(red, green, blue, 255);
    outputImageRGBA[thread_1D_pos] = outputPixel;
}

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{
    checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
    checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
    checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

    size_t filterMemSize = filterWidth * filterWidth * sizeof(float);
    checkCudaErrors(hipMalloc(&d_filter,  filterMemSize));
    checkCudaErrors(hipMemcpy(d_filter, h_filter, filterMemSize, hipMemcpyHostToDevice));
}

void your_gaussian_blur(const uchar4* const h_inputRGBA, 
	            		uchar4* const d_inputRGBA,
                        uchar4* const d_outputRGBA, 
               			const size_t numRows, 
			            const size_t numCols,
                        unsigned char *d_RedBlur, 
                        unsigned char *d_greenBlur, 
                        unsigned char *d_blueBlur,
                        const int filterWidth)
{
    const dim3 threads(BLOCK_COLS, BLOCK_ROWS);
    const dim3 blocks(numCols / threads.x + 1, numRows / threads.y + 1);

    separateChannels<<<blocks, threads>>>(d_inputRGBA, numRows, numCols, d_red, d_green, d_blue);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    gaussianBlur<<<blocks, threads>>>(d_red, d_RedBlur, numRows, numCols, d_filter, filterWidth);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    gaussianBlur<<<blocks, threads>>>(d_green, d_greenBlur, numRows, numCols, d_filter, filterWidth);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    gaussianBlur<<<blocks, threads>>>(d_blue, d_blueBlur, numRows, numCols, d_filter, filterWidth);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    recombineChannels<<<blocks, threads>>>
                    (d_RedBlur, d_greenBlur, d_blueBlur, d_outputRGBA, numRows, numCols);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());
}

void cleanup() 
{
    checkCudaErrors(hipFree(d_red));
    checkCudaErrors(hipFree(d_green));
    checkCudaErrors(hipFree(d_blue));
    checkCudaErrors(hipFree(d_filter));
}
